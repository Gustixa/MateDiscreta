#include "hip/hip_runtime.h"
% % cu
/**
 * --------------------------------------------------------
 * Universidad del Valle de Guatemala
 * CC3056 - Programación de Microprocesadores
 * --------------------------------------------------------
 * Streams.cu
 * --------------------------------------------------------
 * Suma de dos vectores en CUDA
 * Demuestra la forma de usar CUDA 7 Streams para
 * concurrencia simplificada
 * --------------------------------------------------------
 * AUTH.	Mark Harris
 * MOD.   Kimberly B.
 * DATE		2022-10-7
 * --------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SIZE 10
		// GLOBAL1: funcion llamada desde el host y ejecutada en el device (kernel)
		__global__ void
		Kernel1(int *a, int *b, int *c)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	// Solo trabajan N hilos
	if (myID < SIZE)
	{
		c[myID] = a[myID] + b[myID];
	}
}

// GLOBAL2: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void Kernel2(int *d, int *e, int *f)
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	// Solo trabajan N hilos
	if (myID < SIZE)
	{
		f[myID] = d[myID] * e[myID];
	}
}

__global__ void kernel3(int *g, int *h, int *i)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	if (myId < SIZE)
	{
		i[myId] = (g[myId] - h[myId]) * 5;
	}
}

int main(void)
{
	hipStream_t stream1, stream2, stream3;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	int *a1, *b1, *c1; // host vars to use in stream 1 mem ptrs
	int *a2, *b2, *c2; // host vars to use in stream 2 mem ptrs
	int *a3, *b3, *c3;

	int *dev_a1, *dev_b1, *dev_c1; // stream 1 mem ptrs
	int *dev_a2, *dev_b2, *dev_c2; // stream 2 mem ptrs
	int *dev_a3, *dev_b3, *dev_c3;

	// stream 1 - mem allocation at Global memmory for device and host, in order
	hipMalloc((void **)&dev_a1, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b1, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c1, SIZE * sizeof(int));

	hipHostAlloc((void **)&a1, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&b1, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&c1, SIZE * sizeof(int), hipHostMallocDefault);

	// stream 2 - mem allocation at Global memmory for device and host, in order
	hipMalloc((void **)&dev_a2, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b2, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c2, SIZE * sizeof(int));

	hipHostAlloc((void **)&a2, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&b2, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&c2, SIZE * sizeof(int), hipHostMallocDefault);

	// stream 3 - mem allocation at Global memmory for device and host, in order
	hipMalloc((void **)&dev_a3, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b3, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c3, SIZE * sizeof(int));

	hipHostAlloc((void **)&a3, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&b3, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&c3, SIZE * sizeof(int), hipHostMallocDefault);
	// Generate data
	for (int i = 0; i < SIZE; i++)
	{
		a1[i] = 1 + i;
		b1[i] = 5 + i;

		a2[i] = 3 + i;
		b2[i] = 4 + i;

		a3[i] = 7 + i;
		b3[i] = 10 + i;
	}

	for (int i = 0; i < SIZE; i++)
	{
		// STREAM 1
		// HOST TO DEVICE COPY DATA (TO OPERATE)
		hipMemcpyAsync(dev_a1, a1, SIZE * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b1, b1, SIZE * sizeof(int), hipMemcpyHostToDevice, stream1);

		Kernel1<<<1, SIZE, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
		hipMemcpyAsync(c1, dev_c1, SIZE * sizeof(int), hipMemcpyDeviceToHost, stream1);

		// STREAM 2
		// HOST TO DEVICE COPY DATA (TO OPERATE)
		hipMemcpyAsync(dev_a2, a2, SIZE * sizeof(int), hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(dev_b2, b2, SIZE * sizeof(int), hipMemcpyHostToDevice, stream2);

		Kernel2<<<1, SIZE, 1, stream2>>>(dev_a2, dev_b2, dev_c2);
		hipMemcpyAsync(c2, dev_c2, SIZE * sizeof(int), hipMemcpyDeviceToHost, stream2);

		// STREAM 3
		//
		hipMemcpyAsync(dev_a3, a3, SIZE * sizeof(int), hipMemcpyHostToDevice, stream3);
		hipMemcpyAsync(dev_b3, b3, SIZE * sizeof(int), hipMemcpyHostToDevice, stream3);

		Kernel3<<<1, SIZE, 1, stream2>>>(dev_a3, dev_b3, dev_c3);
		hipMemcpyAsync(c3, dev_c3, SIZE * sizeof(int), hipMemcpyDeviceToHost, stream3);
	}

	hipStreamSynchronize(stream1); // wait for stream1 to finish
	hipStreamSynchronize(stream2);
	hipStreamSynchronize(stream3);

	printf("--- STREAM 1 ---\n");
	printf("> Vector a1:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", a1[i]);
	}

	printf("> \n Vector b1:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", b1[i]);
	}

	printf("> \n Vector c1:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", c1[i]);
	}

	printf("\n\n--- STREAM 2 ---\n");

	printf("> Vector a2:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", a2[i]);
	}

	printf("> \n Vector b2:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", b2[i]);
	}

	printf("> \n Vector c2:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", c2[i]);
	}
	printf("\n");

	printf("\n\n--- STREAM 3 ---\n");

	printf("> Vector a3:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", a3[i]);
	}

	printf("> \n Vector b3:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", b3[i]);
	}

	printf("> \n Vector c3:\n");
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", c3[i]);
	}
	printf("\n");
	hipStreamDestroy(stream1); // because we care
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);

	return 0;
}